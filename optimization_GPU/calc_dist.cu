#include "hip/hip_runtime.h"
/*
 * Proj 3-2 SKELETON
 */

#include <float.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "utils.h"
#include <math.h>

/* takes the square root of an INTEGER X and floors it to the nearest int. */
int floor_sqrt(int x) {
    return (int) floor(sqrt((double) x));
}

/* The kernel for horizontal flip function. */
__global__ void flip_kernel(float *arr, int width) {
    
    int N = width * width;

    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;

    int div_val = threadId / width;
    int mod_val = threadId % width;

    int f_threadId = (div_val + 1) * width - 1 - mod_val;

    if (mod_val < width / 2 && f_threadId < N) {
        float temp = arr[threadId];
        arr[threadId] = arr[f_threadId];
        arr[f_threadId] = temp;
    }
}

/* Does a horizontal flip of the array arr */
void flip_horizontal(float *arr, int width) {
    
    int N = width * width;

    int threads_per_block = 512;

    int blocks_per_grid = floor_sqrt(N / threads_per_block)+ 1;

    // create 2-D grid and 1-D block
    dim3 dim_blocks_per_grid(blocks_per_grid, blocks_per_grid);
    dim3 dim_threads_per_block(threads_per_block, 1, 1);

    // launch kernel on GPU
    flip_kernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(arr, width);

    hipDeviceSynchronize();
    CUT_CHECK_ERROR("");
}

/* The kernel for the transpose function. */
__global__ void transpose_kernel(float *arr, int width) {
    
    int N = width * width;

    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;

    int div_val = threadId / width;
    int mod_val = threadId % width;

    int t_threadId = mod_val * width + div_val;

    if (mod_val > div_val && t_threadId < N) {
        float temp = arr[threadId];
        arr[threadId] = arr[t_threadId];
        arr[t_threadId] = temp;
    }
}

/* Transposes the square array ARR. */
void transpose(float *arr, int width) {

    int N = width * width;

    int threads_per_block = 512;

    int blocks_per_grid = floor_sqrt(N / threads_per_block)+ 1;

    dim3 dim_blocks_per_grid(blocks_per_grid, blocks_per_grid);
    dim3 dim_threads_per_block(threads_per_block, 1, 1);

    transpose_kernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(arr, width);

    hipDeviceSynchronize();
    CUT_CHECK_ERROR("");
}

/* The kernel for the rotate function. */
__global__ void rotate_kernel(float *arr, int width) {
    
    int N = width * width;

    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;

    int div_val = threadId / width;
    int mod_val = threadId % width;

    int r_threadId = (width - div_val - 1) * width + mod_val;

    if (div_val < width / 2 && r_threadId < N) {
        float temp = arr[threadId];
        arr[threadId] = arr[r_threadId];
        arr[r_threadId] = temp;
    }
}

/* Rotates the square array ARR by 90 degrees counterclockwise. */
void rotate_ccw_90(float *arr, int width) {
    
    transpose(arr, width);

    int N = width * width;

    int threads_per_block = 512;

    int blocks_per_grid = floor_sqrt(N / threads_per_block)+ 1;

    // create 2-D grid and 1-D block
    dim3 dim_blocks_per_grid(blocks_per_grid, blocks_per_grid);
    dim3 dim_threads_per_block(threads_per_block, 1, 1);

    // launch kernel on GPU
    rotate_kernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(arr, width);

    hipDeviceSynchronize();
    CUT_CHECK_ERROR("");


}

/* The kernel for the distance function. */
__global__ void distance_kernel(float *total, float *temp, float *image, int width, int i_width, int y_off, int x_off) {

    // size of my array
    int N = width * width;
    
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;

    x_off += (threadId % width);
    y_off += (threadId / width);
    int offset_threadId = y_off * i_width + x_off;


    if (threadId < N) {
        total[threadId] = pow((temp[threadId] - image[offset_threadId]), 2);
    }
}

/* Stores the Euclidean distance between template and image in ARR. CPU function where
 * I call distance_kernel. Y_OFF and X_OFF decides the positioning of my TEMP on IMAGE.
 * WIDTH is the width of the template.
 */
void distance(float *arr, float *temp, float *image, int width, int i_width, int y_off, int x_off) {

    // size of my array
    int N = width * width;

    int threads_per_block = 512;

    int blocks_per_grid = floor_sqrt(N / threads_per_block)+ 1;

    // create 2-D grid and 1-D block
    dim3 dim_blocks_per_grid(blocks_per_grid, blocks_per_grid);
    dim3 dim_threads_per_block(threads_per_block, 1, 1);

    // launch kernel on GPU
    distance_kernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(arr, temp, image, width, i_width, y_off, x_off);

    hipDeviceSynchronize();
    CUT_CHECK_ERROR("");

}

/* The kernel for the reduce function. */
__global__ void reduce_kernel(float *arr, int N, int level) {

    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int threadId = blockId * blockDim.x + threadIdx.x;

    if (threadId * level * 2 < N) {
        arr[threadId * (level * 2)] += arr[threadId * (level * 2) + level];
    }

}

/* Stores the sum of the Euclidean distances of each index into the first index of ARR.
 * The array size is LEN.
 */
void reduce(float *arr, int width) {

    int N = width * width;

    int threads_per_block = 512;

    int blocks_per_row = N / (threads_per_block * 2) + 1;

    int level = 1;

    while (level < N) {
        
        dim3 dimGrid(blocks_per_row, 1);
        dim3 dimBlock(threads_per_block, 1, 1);

        reduce_kernel<<<dimGrid, dimBlock>>>(arr, N, level);

        hipDeviceSynchronize();
        CUT_CHECK_ERROR("");

        level *= 2;

        blocks_per_row /= 2;
        if (blocks_per_row == 0) {
            blocks_per_row = 1;
        }
    }

}

/* Returns the squared Euclidean distance between TEMPLATE and IMAGE. The size of IMAGE
 * is I_WIDTH * I_HEIGHT, while TEMPLATE is square with side length T_WIDTH. The template
 * image should be flipped, rotated, and translated across IMAGE.
 */
float calc_min_dist(float *image, int i_width, int i_height, float *temp, int t_width) {
    // float* image and float* temp are pointers to GPU addressible memory
    // You MAY NOT copy this data back to CPU addressible memory and you MAY 
    // NOT perform any computation using values from image or temp on the CPU.
    // The only computation you may perform on the CPU directly derived from distance
    // values is selecting the minimum distance value given a calculated distance and a 
    // "min so far"

    // host holds the current lowest distance.
    float min_dist = FLT_MAX;
    float calc_dist = 0;

    // length of template
    int N = t_width * t_width;

    // curr_dist stores the resulting Euclidean distance of each index in GPU. 
    float *curr_dist;
    hipMalloc(&curr_dist, N * sizeof(float));

    // calculate the offset for translation.
	int x_offset = i_width - t_width + 1;
	int y_offset = i_height - t_width + 1;

    // scan through the image
	for (int flip = 0; flip < 2; flip ++) {
		for (int rotate = 0; rotate < 4; rotate++) {
			for (int y_off = 0; y_off < y_offset; y_off++) {
				for (int x_off = 0; x_off < x_offset; x_off++) {

                    distance(curr_dist, temp, image, t_width, i_width, y_off, x_off);

                    reduce(curr_dist, t_width);

                    // copy the first index of curr_dist into calc_dist
                    hipMemcpy(&calc_dist, curr_dist, sizeof(float), hipMemcpyDeviceToHost);

					if (calc_dist < min_dist) {
						min_dist = calc_dist;
					}
					calc_dist = 0;
				}
			}
			rotate_ccw_90(temp, t_width);
		}
		flip_horizontal(temp, t_width);
	}
    hipFree(curr_dist);
    return min_dist;
}
