#include "hip/hip_runtime.h"
/*
 * Proj 3-2 SKELETON
 */

#include <float.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "utils.h"

/* Does a horizontal flip of the array arr */
void flip_horizontal(float *arr, int width) {
	for (int i = 0; i < width; i++) {
	    for (int k = 0; k < width/2; k++) {
			swap(&arr[width * i + k], &arr[width * i + width - 1 - k]);
		}
	}	
}

/* Transposes the square array ARR. */
void transpose(float *arr, int width) {
	int offset = 1;
	for (int y = 0; y < width/2 + 1; y++) {
		for (int x = offset; x < width; x++) {
			swap(&arr[y * width + x], &arr[x * width + offset - 1]);
		}
		offset += 1;
	}
}

/* Rotates the square array ARR by 90 degrees counterclockwise. */
void rotate_ccw_90(float *arr, int width) {
	transpose(arr, width);
	for (int y = 0; y < width/2; y++) {
		for (int x = 0; x < width; x++) {
			swap(&arr[y * width + x], &arr[width * (width - (1 + y)) + x]);
		}
	}
}

/* The kernel for the calc_min_dist function. It takes in a TEMPLATE and an IMAGE
 * and calculates the euclidean distance between the two. It then puts the answer
 * into TOTAL.
 */
__global__ void calc_min_dist_kernel(float *total, float *temp, float *image, int width) {
    int thisThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (thisThreadIndex < width * width) {
        total[0] += pow(temp[thisThreadIndex] - image[thisThreadIndex], 2);
    }
}

/* Returns the squared Euclidean distance between TEMPLATE and IMAGE. The size of IMAGE
 * is I_WIDTH * I_HEIGHT, while TEMPLATE is square with side length T_WIDTH. The template
 * image should be flipped, rotated, and translated across IMAGE.
 */
float calc_min_dist(float *image, int i_width, int i_height, float *temp, int t_width) {
    // float* image and float* temp are pointers to GPU addressible memory
    // You MAY NOT copy this data back to CPU addressible memory and you MAY
    // NOT perform any computation using values from image or temp on the CPU.

    float min_dist = FLT_MAX;
    float *curr_dist;
    hipMalloc(&curr_dist, 1 * sizeof(float));
	int x_offset = i_width - t_width + 1;
	int y_offset = i_height - t_width + 1;
	int flip = 0;

    int threads_per_block = 512; // set to the max possible

    // set an initial number of blocks_per_grid
    int blocks_per_grid = ((t_width * t_width) / threads_per_block) + 1;  

	while (flip < 2) {
		for (int rotate = 0; rotate < 4; rotate++) {
			for (int y_off = 0; y_off < y_offset; y_off++) {
				for (int x_off = 0; x_off < x_offset; x_off++) {

                    printf("hello");
                    // create 3-dim vector objects to initialize values
                    dim3 dim_blocks_per_grid(blocks_per_grid, 1);
                    dim3 dim_threads_per_block(threads_per_block, 1, 1);

                    // launch kernel on GPU
                    calc_min_dist_kernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(curr_dist, temp, image, t_width);

                    // wait for GPU to finish computation
                    hipDeviceSynchronize();

                    printf("a");
                    // debugging sanity.
                    CUT_CHECK_ERROR("");
                    
                    printf("b");
					if (curr_dist[0] < min_dist) {
						min_dist = curr_dist[0];
					}
					curr_dist[0] = 0;
				}
			}
			rotate_ccw_90(temp, t_width);
		}
		flip_horizontal(temp, t_width);
		flip++;
	}
    return min_dist;
}   
