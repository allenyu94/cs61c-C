#include "hip/hip_runtime.h"
/*
 * Proj 3-2 SKELETON
 */

#include <float.h>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "utils.h"

/* kernel for horizontal flip on GPU. */
__global__ void flip_horizontal_kernel(float *arr, int width) {
    int thisThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int div_val = thisThreadIndex / width;
    int mod_val = thisThreadIndex % width;

    if (mod_val < width / 2 && thisThreadIndex < (width * width)) {
        float temp = arr[thisThreadIndex];
        arr[thisThreadIndex] = arr[(div_val + 1) * width - (mod_val + 1)];
        arr[(div_val + 1) * width - (mod_val + 1)] = temp;
    }
}

/* Does a horizontal flip of the array arr */
void flip_horizontal(float *arr, int width) {
    int threads_per_block = 512;
    int blocks_per_grid = (width * width / threads_per_block) + 1;

    dim3 dim_blocks_per_grid(blocks_per_grid, 1);
    dim3 dim_threads_per_block(threads_per_block, 1, 1);

    flip_horizontal_kernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(arr, width);

    hipDeviceSynchronize();

    CUT_CHECK_ERROR("");
}

/* kernel for transpose on GPU. */
__global__ void transpose_kernel(float *arr, int width) {
    int thisThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int div_val = thisThreadIndex / width;
    int mod_val = thisThreadIndex % width;
    
    if (thisThreadIndex < (width * width) && thisThreadIndex % (width + 1) != 0) {
        float temp = arr[thisThreadIndex];
        arr[thisThreadIndex] = arr[mod_val * width + div_val];
        arr[mod_val * width + div_val] = temp;
    }

}

/* Transposes the square array ARR. */
void transpose(float *arr, int width) {
    int threads_per_block = 512;
    int blocks_per_grid = (width * width / threads_per_block) + 1;

    dim3 dim_blocks_per_grid(blocks_per_grid, 1);
    dim3 dim_threads_per_block(threads_per_block, 1, 1);

    transpose_kernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(arr, width);

    hipDeviceSynchronize();

    CUT_CHECK_ERROR("");
}

/* kernel for rotation on GPU. */
__global__ void rotate_kernel(float *arr, int width) {
    int thisThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int div_val = thisThreadIndex / width;
    int mod_val = thisThreadIndex % width;
    
    if (thisThreadIndex < (width * width) && div_val > (width / 2)) {
        float temp = arr[thisThreadIndex];
        arr[thisThreadIndex] = arr[(width - div_val - 1) * width + mod_val];
        arr[(width - div_val - 1) * width + mod_val] = temp;
    }
}

/* Rotates the square array ARR by 90 degrees counterclockwise. */
void rotate_ccw_90(float *arr, int width) {
    int threads_per_block = 512;
    int blocks_per_grid = (width * width / threads_per_block) + 1;

    dim3 dim_blocks_per_grid(blocks_per_grid, 1);
    dim3 dim_threads_per_block(threads_per_block, 1, 1);

    rotate_kernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(arr, width);

    hipDeviceSynchronize();

    CUT_CHECK_ERROR("");
}

/* The kernel for the calc_min_dist function. It takes in a TEMPLATE and an IMAGE
 * and calculates the euclidean distance between the two. It then puts the answer
 * into TOTAL.
 */
__global__ void calc_min_dist_kernel(float *total, float *temp, float *image, int width) {
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;

    if (threadId < (width * width)) {
        total[threadId] = pow((temp[threadId] - image[threadId]), 2);
    }
}

__global__ void add_distance_kernel(float *arr, int len, int level) {
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;

    if (threadId * level * 2 < len) {
        arr[threadId * level * 2] += arr[threadId * level * 2 + level];
    }

}

/* Returns the squared Euclidean distance between TEMPLATE and IMAGE. The size of IMAGE
 * is I_WIDTH * I_HEIGHT, while TEMPLATE is square with side length T_WIDTH. The template
 * image should be flipped, rotated, and translated across IMAGE.
 */
float calc_min_dist(float *image, int i_width, int i_height, float *temp, int t_width) {
    // float* image and float* temp are pointers to GPU addressible memory
    // You MAY NOT copy this data back to CPU addressible memory and you MAY 
    // NOT perform any computation using values from image or temp on the CPU.

    int len = t_width * t_width;

    // host copies of min_dist, curr_dist_temp
    float min_dist = FLT_MAX;
    //float *curr_dist_temp;
    //curr_dist_temp = (float *) malloc(len * sizeof(float));

    // device copies of curr_dist
    float *curr_dist;
    hipMalloc(&curr_dist, len * sizeof(float));

    // instantiating the grid and block dimensions to 2D Grid and 1D block
    int threads_per_block = 512;

    int blocks_per_grid = (t_width / threads_per_block) + 1; 

    dim3 dim_blocks_per_grid(blocks_per_grid, blocks_per_grid); // gridDim.x = blocks_per_grid_x and gridDim.y = blocks_per_grid_y
    dim3 dim_threads_per_block(threads_per_block, 1, 1); // blockDim.x = threads_per_block

    // Launch calc_min_dist_kernel on GPU
    calc_min_dist_kernel<<<dim_blocks_per_grid, dim_threads_per_block>>>(curr_dist, temp, image, t_width);

    // Wait for GPU to finish computation
    hipDeviceSynchronize();

    CUT_CHECK_ERROR("");

    // Copy result back to host
    // hipMemcpy(curr_dist_temp, curr_dist, t_width * t_width * sizeof(float), hipMemcpyDeviceToHost);

    
    int level = 1;
    while (level != len) {
        
        int threads_per_block2 = 512;

        int blocks_per_grid2 = (t_width / threads_per_block2) + 1; 
    
        dim3 dim_blocks_per_grid2(blocks_per_grid2, blocks_per_grid2); // gridDim.x = blocks_per_grid_x and gridDim.y = blocks_per_grid_y
        dim3 dim_threads_per_block2(threads_per_block, 1, 1); // blockDim.x = threads_per_block

        add_distance_kernel<<<dim_blocks_per_grid2, dim_threads_per_block2>>>(curr_dist, t_width * t_width, level);

        hipDeviceSynchronize();

        CUT_CHECK_ERROR("");

        level *= 2;

        blocks_per_grid2 = ((sqrt(len / 2) + 1) / threads_per_block2);
        if (blocks_per_grid2 == 0) {
            blocks_per_grid2 = 1;
        }
        
    } 

    float curr_min = 0.0;

    hipMemcpy(&curr_min, curr_dist, sizeof(float), hipMemcpyDeviceToHost);

    if (curr_min < min_dist) {
        min_dist = curr_min;
    }

    // Cleanup
    hipFree(curr_dist);

    return min_dist;
    
}

